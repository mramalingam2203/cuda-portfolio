#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "montecarlo.cuh"

#define THREADS_PER_BLOCK 256

int main() {
    int n = 1 << 18;    // ~262,144 simulations
    int steps = 252;    // trading days in a year
    float S0 = 100.0f, mu = 0.05f, sigma = 0.2f, T = 1.0f;

    size_t resultsSize = n * sizeof(float);
    size_t pathsSize = n * steps * sizeof(float);

    float *d_results;
    hipMalloc(&d_results, resultsSize);

    float *d_paths;
    hipMalloc(&d_paths, pathsSize);

    hiprandStatePhilox4_32_10_t *d_state;
    hipMalloc(&d_state, n * sizeof(hiprandStatePhilox4_32_10_t));

    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Initialize RNG states
    initCurandPhilox<<<blocks, THREADS_PER_BLOCK>>>(d_state, time(NULL), n);
    hipDeviceSynchronize();

    // Run multi-step GBM simulation
    gbmPathsMultiStep<<<blocks, THREADS_PER_BLOCK>>>(
        d_state, d_results, d_paths,
        n, steps, S0, mu, sigma, T, true
    );
    hipDeviceSynchronize();

    // Copy back final prices
    std::vector<float> h_results(n);
    hipMemcpy(h_results.data(), d_results, resultsSize, hipMemcpyDeviceToHost);

    // Compute average price
    long double sum = 0.0;
    for (auto &val : h_results) sum += val;
    std::cout << "Estimated final price (mean): " << sum / n << std::endl;

    hipFree(d_results);
    hipFree(d_paths);
    hipFree(d_state);

    return 0;
}
